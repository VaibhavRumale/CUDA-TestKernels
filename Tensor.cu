#include "hip/hip_runtime.h"
// Compiled with: nvcc -o tensor_core tensor_core.cu -arch=sm_70

#include <hip/hip_fp16.h>

// Kernel for FP16 Tensor Core matrix multiplication
__global__ void tcMul(const __half* A, const __half* B, __half* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __half result = __float2half(0.0f);
    for (int k = 0; k < K; ++k) {
        __half a = __ldg(&A[row * K + k]); // Use __ldg to load from global memory
        __half b = __ldg(&B[k * N + col]);
        result = __hfma(result, a, b, result); // Fused multiply-add with Tensor Cores
    }
    C[row * N + col] = result;
}

int main() {
    const int M = 1024;
    const int N = 1024; 
    const int K = 1024;

    // Allocate host memory
    __half* h_A = (__half*)malloc(M * K * sizeof(__half));
    __half* h_B = (__half*)malloc(K * N * sizeof(__half));
    __half* h_C = (__half*)malloc(M * N * sizeof(__half));

    // Initialize input matrices
    // ...

    // Allocate device memory
    __half* d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(__half));
    hipMalloc(&d_B, K * N * sizeof(__half));
    hipMalloc(&d_C, M * N * sizeof(__half));

    // Copy input data to device
    hipMemcpy(d_A, h_A, M * K * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(__half), hipMemcpyHostToDevice);

    // Launch Tensor Core kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
    tcMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);

    // Copy output data back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(__half), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B); 
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
