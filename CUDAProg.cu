#include "hip/hip_runtime.h"
// --atomics 15 ---fake_divergence -g 42,55,3 -l 6,1,1
#ifndef NO_GROUP_DIVERGENCE
#define GROUP_DIVERGE(a, b) get_block_id(a)
#else
#define GROUP_DIVERGE(x, y) (y)
#endif

#ifndef NO_FAKE_DIVERGENCE
#define FAKE_DIVERGE(x, y, z) (x - y)
#else
#define FAKE_DIVERGE(x, y, z) (z)
#endif


// Seed: 1123171451

#include "CUDA.h"

/* --- Struct/Union Declarations --- */
/* --- Struct/Union Declarations End --- */
struct S1 {
    int32_t g_3;
    uint64_t global_0_offset;
    uint64_t global_1_offset;
    uint64_t global_2_offset;
    uint64_t local_0_offset;
    uint64_t local_1_offset;
    uint64_t local_2_offset;
    uint64_t group_0_offset;
    uint64_t group_1_offset;
    uint64_t group_2_offset;
};


/* --- FORWARD DECLARATIONS --- */
__device__ int32_t  func_1(struct S1 * p_4);


/* --- FUNCTIONS --- */
/* ------------------------------------------ */
/* 
 * reads : p_4->g_3
 * writes: p_4->g_3
 */
__device__ int32_t  func_1(struct S1 * p_4)
{ /* block id: 4 */
    int32_t *l_2 = &p_4->g_3;
    (*l_2) |= 1L;
    return (*l_2);
}


extern "C" __global__ void entry( long *result,  volatile uint *g_atomic_input,  volatile uint *g_special_values , int *sequence_input) {
    int ;
    struct S1 c_5;
    struct S1* p_4 = &c_5;
    struct S1 c_6 = {
        0x5AE8C9F3L, // p_4->g_3
        sequence_input[get_global_id(0)], // p_4->global_0_offset
        sequence_input[get_global_id(1)], // p_4->global_1_offset
        sequence_input[get_global_id(2)], // p_4->global_2_offset
        sequence_input[get_local_id(0)], // p_4->local_0_offset
        sequence_input[get_local_id(1)], // p_4->local_1_offset
        sequence_input[get_local_id(2)], // p_4->local_2_offset
        sequence_input[get_group_id(0)], // p_4->group_0_offset
        sequence_input[get_group_id(1)], // p_4->group_1_offset
        sequence_input[get_group_id(2)], // p_4->group_2_offset
    };
    c_5 = c_6;
    __syncthreads();
    func_1(p_4);
    __syncthreads();
    uint64_t crc64_context = 0xFFFFFFFFFFFFFFFFUL;
    int print_hash_value = 0;
    transparent_crc(p_4->g_3, "p_4->g_3", print_hash_value);
       result[get_linear_global_id()] = crc64_context ^ 0xFFFFFFFFFFFFFFFFUL;
}
